#include "hip/hip_runtime.h"
#include "GPUrender.cuh"
//������Ʈ 20, �ڵ����� 12


//gpu ����޸�
__constant__ int gResolution;
__constant__ int gVolumeSize[3];
__constant__ int gBlockSize[3];//volume�� 8ĭ�� ���� x, y ����
__constant__ float gEye[3];
__constant__ float gDir[3];
__constant__ float gBackDir[3];
__constant__ float gCross[3];
__constant__ float gU[3];
__constant__ float gL[3];
__constant__ int gValidDir[3];
//__constant__ bool emptyBlock[32*32*29];
__constant__ bool emptyBlock[40*40*40]; //������ Ŀ���� 40���� ũ�� ����?

hipArray* hipArray = {0};
GLuint pbo = 0;     // EyeBody pixel buffer object
struct hipGraphicsResource *cuda_pbo_resource;
//gpu �ؽ��ĸ޸�
texture<unsigned char, 3, hipReadModeNormalizedFloat> texPtr;

__inline__ __host__ __device__ void vec_add(float a[3], float b[3], float c[3]){
	c[0] = a[0] + b[0];
	c[1] = a[1] + b[1];
	c[2] = a[2] + b[2];
}
__inline__ __host__ __device__ void vec_sub(float a[3], float b[3], float c[3]){
	c[0] = a[0] - b[0];
	c[1] = a[1] - b[1];
	c[2] = a[2] - b[2];
}
__inline__ __host__ __device__ void s_product(float a[3], float size, float b[3]){
	b[0] = a[0] * size;
	b[1] = a[1] * size;
	b[2] = a[2] * size;
}
__inline__ __host__ __device__ void cross_product(float a[3], float b[3], float c[3])
{
	c[0] = a[1]*b[2] - a[2]*b[1];
	c[1] = a[2]*b[0] - a[0]*b[2];
	c[2] = a[0]*b[1] - a[1]*b[0];
}
__inline__ __host__ __device__ float vec_lenth(float a[3]){
	return sqrtf(a[0]*a[0] + a[1]*a[1] + a[2]*a[2]);
}
__inline__ __host__ __device__ float inner_product(float a[3], float b[3]){
	float buf = a[0]*b[0];
	buf += a[1]*b[1];
	buf += a[2]*b[2];
	return buf;
}
__inline__ __device__ void getNormal(float pos[3], float N[3]){
	N[0] = (tex3D(texPtr, pos[0]+1, pos[1], pos[2]) - tex3D(texPtr, pos[0]-1, pos[1], pos[2]))/2.0f;
	N[1] = (tex3D(texPtr, pos[0], pos[1]+1, pos[2]) - tex3D(texPtr, pos[0], pos[1]-1, pos[2]))/2.0f;
	N[2] = (tex3D(texPtr, pos[0], pos[1], pos[2]+1) - tex3D(texPtr, pos[0], pos[1], pos[2]-1))/2.0f;


	float len = vec_lenth(N);
	if(len != 0)//0���� �������°� ����
		s_product(N, 1/vec_lenth(N), N); //�������ͷ� �����
}
__inline__ __device__ float sign(float a){
	if(a > 0)
		return 1.0f;
	if(a < 0)
		return -1.0f;
	return 0.0f;
}

//parallel �Լ�
__inline__ __device__ bool IsIntersectRayBox1(float& startT, float& endT, float pos[3], int tx, int ty){
	float buf[3];
	float start[3];
	float dx[3], dy[3];
	float delta[3];
	float Max[3], Min[3];//x, y, z�� �ִ�, �ּ�
	int j = 0;

	s_product(gCross, tx-gResolution*0.5f, dx);//x���
	s_product(gU, ty-gResolution*0.5f, dy);//y���
	vec_add(dx, dy, delta);//x+y = point
	vec_add(gEye, delta, start);//start+eye = start <- ��������

	for(int i = 0; i < 3; i++){
		float a, b;
		if(gValidDir[i] == 1){
			a = (gVolumeSize[i]-1 - start[i])/ gDir[i];
			b = (0.0f - start[i])/ gDir[i];
			

			if(a > b){//ũ�� ����
				Max[j] = a;
				Min[j] = b;
			}
			else{
				Max[j] = b;
				Min[j] = a;
			}
			j++;
		}
	}

	startT = Min[0];
	for(int i = 1; i < j; i++){//Min�� Max�� ã��. = startT�� �ȴ�.
		if(startT < Min[i])
			startT = Min[i];
	}
	startT += 0.001f;

	endT = Max[0];
	for(int i = 1; i < j; i++){//Max�� Min�� ã��. - endT�� �ȴ�.
		if(endT > Max[i])
			endT = Max[i];
	}
	endT -= 0.001f;

	//������ �����Ͱ� ������ ���� ã�ƺ���.
	s_product(gDir, startT, buf);
	vec_add(start, buf, pos);

	//������ �ڽ��� ������ ã���� ������
	float maxBox[3] = {gVolumeSize[0], gVolumeSize[1], gVolumeSize[2]};
	float minBox[3] = {0.0f, 0.0f, 0.0f};

	float result1[3];
	float result2[3];

	for(int i = 0; i < 3; i++){
		result1[i] = sign(minBox[i]- pos[i]);
		result2[i] = sign(pos[i] - maxBox[i]);
	}
	float k = inner_product(result1, result2);
	if(k == 3.0f)
		return true;
	return false;

}
__inline__ __device__ int EmptySpaceLeap1(float pos[3]){
	//���� �ڽ��� ��������� Ȯ���ϸ� �����ڽ��� �����Ѵ�.
	int dt = 0;
	float currentBox[3] = {floorf(pos[0]*0.125f), floorf(pos[1]*0.125f), floorf(pos[2]*0.125f)};
	int currentBoxId = currentBox[0]+ currentBox[1]*gBlockSize[0] + currentBox[2]*gBlockSize[0]*gBlockSize[1];

	if(emptyBlock[currentBoxId]){
		while(true){
			dt++;
			vec_add(pos, gDir, pos);

			float forwardBox[3] = {floorf(pos[0]*0.125f), floorf(pos[1]*0.125f), floorf(pos[2]*0.125f)};
			int forwardBoxId = forwardBox[0]+ forwardBox[1]*gBlockSize[0] + forwardBox[2]*gBlockSize[0]*gBlockSize[2];

			//���ο� �ڽ��� �����ϸ� ���������˻縦 ������.
			if(currentBoxId != forwardBoxId)
				return dt;									
		}
	}	
	//������������� ���ĺ���
	return dt;
}
__inline__ __device__ float AlphaBlending1(float4* PIT, float pos[3], float3& cAcc, const float aOld){
	unsigned char nowData = (unsigned char)(tex3D(texPtr, pos[0], pos[1], pos[2])*255.0f);		
	unsigned char nextData = (unsigned char)(tex3D(texPtr, pos[0]+gDir[0], pos[1]+gDir[1], pos[2]+gDir[2])*255.0f);
	
	if((nowData + nextData) == 0)
		return aOld;

	float N[3];// �ȼ��� ��������
	getNormal(pos, N);//�������͸� ã�´�.

	float NL = fabs(inner_product(N, gL));//N�� L�� ����  - ���밪
	float NH = fabs(pow(inner_product(N, gDir), 16));
	float light = 0.2f + 0.7f*NL + 0.1f*NH; 
	if(light > 1.0f)
		light = 1.0f;

	int index = nowData*256 + nextData;
	light *= 1.0f-aOld;
	float alpha = PIT[index].w;
	cAcc.x += PIT[index].x*light;
	cAcc.y += PIT[index].y*light;
	cAcc.z += PIT[index].z*light;
	
	return 1.0f-(1.0f-aOld) * (1.0f-alpha);//���İ� ����
}
__inline__ __device__ float3 RayTracing1(float4* PIT, float start[3], const float startT, const float endT){
	float pos[3] = {start[0], start[1], start[2]};
	float aNew = 0.0f;
	float aOld = 0.0f;
	float3 cAcc = {0};
	
	for(float t = startT; t <= endT; t+=1.0f){
		int dt = EmptySpaceLeap1(pos);

		//dt�� ������ ĭ�� 
		if(dt){//������ ����ִٸ�			
			t+=dt-1.0f;
			continue;
		}		

		aNew = AlphaBlending1(PIT, pos, cAcc, aOld);		

		//Early Ray Termination
		if(aNew > 0.99f)
			break;	

		aOld = aNew;
		vec_add(pos, gDir, pos);
	}
	return cAcc;
}
__global__ void G_Parallel(unsigned char* tex, float4* PIT){
	//const int tx = blockDim.x*blockIdx.x + threadIdx.x;//������ x��ǥ
	//const int ty = blockDim.y*blockIdx.y + threadIdx.y;//������ y��ǥ
	//const int locTexture = ty*256 + tx;//���������� ���� ��ǥ
	const int locTexture = blockDim.x*blockIdx.x + threadIdx.x;
	const int ty = locTexture/gResolution;
	const int tx = locTexture%gResolution;

	float pos[3];
	float startT, endT;
	//IsIntersectRayBox�� ������ ��ȿ���� Ȯ���ϰ� ����T�� ��T�� ���Ѵ�.
	if(!IsIntersectRayBox1(startT, endT, pos, tx, ty)){
		tex[locTexture*3] = 0;
		tex[locTexture*3 + 1] = 0;
		tex[locTexture*3 + 2] = 0;
		return;//��ȿ���� ���� ��ǥ�� ������ ����
	}

	float3 cAcc = RayTracing1(PIT, pos, startT, endT);	

	tex[locTexture*3] = (unsigned char)(cAcc.x*255.0f);
	tex[locTexture*3 + 1] = (unsigned char)(cAcc.y*255.0f);
	tex[locTexture*3 + 2] = (unsigned char)(cAcc.z*255.0f);
}
//parallel �Լ� ��

//perspective �Լ�
__inline__ __device__ bool IsIntersectRayBox2(float& startT, float& endT, float pos[3], float dir[3], int tx, int ty){
	float buf[3];
	float f[3];
	float cameraCenter[3];
	float start[3];
	float Max[3], Min[3];//x, y, z�� �ִ�, �ּ�
	float dx[3], dy[3];
	float delta[3];
	int j = 0;

	s_product(gCross, tx-gResolution*0.5f, dx);//x���
	s_product(dx, 0.005f, dx);
	s_product(gU, ty-gResolution*0.5f, dy);//y���
	s_product(dy, 0.005f, dy);
	vec_add(dx, dy, delta);//dx+dy = delta

	s_product(gDir, 1.0f, f);
	vec_add(gEye, f, cameraCenter);
	vec_add(cameraCenter, delta, start);
	vec_sub(start, gEye, dir);//�� �ȼ����� �ٸ� dir�� ������.
	s_product(dir, 1/vec_lenth(dir), dir);

	for(int i = 0; i < 3; i++){
		float a, b;
		if(gValidDir[i] == 1){			
			a = (gVolumeSize[i]-1 - start[i])/ dir[i];
			b = (0.0f - start[i])/ dir[i];
			

			if(a > b){//ũ�� ����
				Max[j] = a;
				Min[j] = b;
			}
			else{
				Max[j] = b;
				Min[j] = a;
			}
			j++;
		}
	}

	endT = Max[0];
	for(int i = 1; i < j; i++){//Max�� Min�� ã��. - endT�� �ȴ�.
		if(endT > Max[i])
			endT = Max[i];
	}
	endT -= 0.001f;

	//����ũ��
	float maxBox[3] = {gVolumeSize[0], gVolumeSize[1], gVolumeSize[2]};
	float minBox[3] = {0.0f, 0.0f, 0.0f};
	float result1[3];
	float result2[3];
	float k;

	for(int i = 0; i < 3; i++){
		result1[i] = sign(minBox[i]- start[i]);
		result2[i] = sign(start[i] - maxBox[i]);
	}
	k = inner_product(result1, result2);

	//start�� ���� ���ο� �ִ��� Ȯ�����Ŀ� �����̸� �������� �˻����ʿ䰡 ����.
	if(k == 3.0f){//�ڽ�����
		pos[0] = start[0];
		pos[1] = start[1];
		pos[2] = start[2];
		startT = 0.0f;
		return true;
	}

	//���� �ܺο� ������� �������� ã�ƾ��Ѵ�.
	startT = Min[0];
	for(int i = 1; i < j; i++){//Min�� Max�� ã��. = startT�� �ȴ�.
		if(startT < Min[i])
			startT = Min[i];
	}
	startT += 0.001f;


	//���� �ܺο� ������쿡�� ������ �ڽ��� ������ ������ ã�ƾ���
	s_product(dir, startT, buf);
	vec_add(start, buf, pos);

	for(int i = 0; i < 3; i++){
		result1[i] = sign(minBox[i]- pos[i]);
		result2[i] = sign(pos[i] - maxBox[i]);
	}
	k = inner_product(result1, result2);

	if(k == 3.0f)//�ڽ� ������
		return true;
	return false;
}
__inline__ __device__ int EmptySpaceLeap2(float pos[3], float dir[3]){
	//���� �ڽ��� ��������� Ȯ���ϸ� �����ڽ��� �����Ѵ�.
	int dt = 0;
	float currentBox[3] = {floorf(pos[0]*0.125f), floorf(pos[1]*0.125f), floorf(pos[2]*0.125f)};
	float currentBoxId = currentBox[0]+ currentBox[1]*gBlockSize[0] + currentBox[2]*gBlockSize[0]*gBlockSize[1];

	if(emptyBlock[(int)currentBoxId]){
		while(true){
			dt++;
			vec_add(pos, dir, pos);

			float forwardBox[3] = {floorf(pos[0]*0.125f), floorf(pos[1]*0.125f), floorf(pos[2]*0.125f)};
			float forwardBoxId = forwardBox[0]+ forwardBox[1]*gBlockSize[0] + forwardBox[2]*gBlockSize[0]*gBlockSize[2];

			//���ο� �ڽ��� �����ϸ� ���������˻縦 ������.
			if(currentBoxId != forwardBoxId)
				break;						
		}
	}	
	//������������� ���ĺ���
	return dt;
}
__inline__ __device__ float AlphaBlending2(float4* PIT, float pos[3], float dir[3], float3& cAcc, const float aOld){
	unsigned char nowData = (unsigned char)(tex3D(texPtr, pos[0], pos[1], pos[2])*255.0f);
	unsigned char nextData = (unsigned char)(tex3D(texPtr, pos[0]+dir[0], pos[1]+dir[1], pos[2]+dir[2])*255.0f);

	float N[3];// �ȼ��� ��������
	getNormal(pos, N);//�������͸� ã�´�.

	float NL = fabs(inner_product(N, gL));//N�� L�� ����  - ���밪
	float NH = fabs(pow(inner_product(N, gDir), 16));
	float light = 0.2f + 0.7f*NL + 0.1f*NH; 
	if(light > 1.0f)
		light = 1.0f;

	int index = nowData*256 + nextData;
	light *= 1.0f-aOld;
	float alpha = PIT[index].w;
	cAcc.x += PIT[index].x*light;
	cAcc.y += PIT[index].y*light;
	cAcc.z += PIT[index].z*light;
	
	return 1.0f-(1.0f-aOld) * (1.0f-alpha);//���İ� ����
}
__inline__ __device__ float3 RayTracing2(float4* PIT, float start[3], float dir[3], const float startT, const float endT){
	float pos[3] = {start[0], start[1], start[2]};
	float aNew = 0.0f;
	float aOld = 0.0f;
	float3 cAcc = {0};

	for(float t = startT; t <= endT; t+=1.0f){
		int dt = EmptySpaceLeap2(pos, dir);

		//����ִٸ�
		if(dt){
			t+=dt-1.0f;
			continue;
		}		

		//������� �ƴ϶��
		aNew = AlphaBlending2(PIT, pos, dir, cAcc, aOld);		

		//Early Ray Termination
		if(aNew > 0.99f)
			break;		
		aOld = aNew;

		vec_add(pos, dir, pos);
	}
	return cAcc;
}
__global__ void G_Perspective(unsigned char* tex, float4* PIT){
	const int locTexture = blockDim.x*blockIdx.x + threadIdx.x;
	const int ty = locTexture/gResolution;
	const int tx = locTexture%gResolution;

	float pos[3];
	float dir[3];
	float startT, endT;
	//IsIntersectRayBox�� ������ ��ȿ���� Ȯ���ϰ� ����T�� ��T�� ���Ѵ�.
	if(!IsIntersectRayBox2(startT, endT, pos, dir, tx, ty)){
		tex[locTexture*3] = 0;
		tex[locTexture*3 + 1] = 0;
		tex[locTexture*3 + 2] = 0;
		return;//��ȿ���� ���� ��ǥ�� ������ ����
	}

	float3 cAcc = RayTracing2(PIT, pos, dir, startT, endT);	
	tex[locTexture*3] = (unsigned char)(cAcc.x*255.0f);
	tex[locTexture*3 + 1] = (unsigned char)(cAcc.y*255.0f);
	tex[locTexture*3 + 2] = (unsigned char)(cAcc.z*255.0f);
}
//perspective �Լ� ��

__global__ void ChangeAlpha(float* alphaTable, int* transparentTable, int* aSAT){
	int i = threadIdx.x;

	if(alphaTable[i] == 0)
		transparentTable[i] = 0;
	else		
		transparentTable[i] = 1;
	
	__syncthreads();
	for(int j = i; j < 256; j++)
		atomicAdd(&aSAT[j+1], transparentTable[i]);
	if(i == 0)
		aSAT[0] = 0;
}
__global__ void InitMinMaxEmptyBlock(unsigned char* emptyBlockMax, unsigned char* emptyBlockMin){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int bz = i/(gBlockSize[0]*gBlockSize[1]);
	int by = (i%(int)(gBlockSize[0]*gBlockSize[1]))/gBlockSize[0];
	int bx = i%(int)gBlockSize[1];


	float m = 1.0f;
	float M = 0.0f;
	int vz, vy, vx;
	for(vz = bz*8; vz <= bz*8 + 8; vz++){
		if(vz > gVolumeSize[2]-1)
			break;
		for(vy = by*8; vy <= by*8 + 8; vy++){  
			for(vx = bx*8; vx <= bx*8 + 8; vx++){         
				float data = tex3D(texPtr, vx, vy, vz);
				M = max(M, data);
				m = min(m, data);
			}
		}
	}

	emptyBlockMin[i] = (unsigned char)(m*255);
	emptyBlockMax[i] = (unsigned char)(M*255);
}
__global__ void InitEmptyBlock(bool* emptyBlock, unsigned char* emptyBlockMax, unsigned char* emptyBlockMin, int* aSAT){
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	int *pSAT = &(aSAT[1]);

	emptyBlock[i] = (pSAT[emptyBlockMax[i]] == pSAT[emptyBlockMin[i]-1]) ? true : false;
}
__global__ void InitPreIntegration(float4* pit, float* alphaTable, float3* colorTable){
	int i = threadIdx.x;

	if(i == 255){
		for(int j = 0; j < 256; j++){
			float A = alphaTable[j];
			int index = j*256+j;
			pit[index].w = A;
			pit[index].x = colorTable[j].x * A;
			pit[index].y = colorTable[j].y * A;
			pit[index].z = colorTable[j].z * A;
		}
		return;
	}
	

	int k = 255 - i;
	float samplingTable[256];
	for(int j = 0; j < 256; j++){
		float A = alphaTable[j];//�ּҰ�+j��°�� alpha���� ã�´�. s == e�� �������ʿ����
		A = 1.0f - pow(1.0f - A, 1.0f/k);
		samplingTable[j] = A;
	}
	int e = 255-i;
	int s = 0;
	for( ; s <= i; s++,e++){
		float A = 0, aOld = 0, aNew = 0;
		float3 cAcc = {0};
		//���簢���� �밢�� /�� �������� ���ʻ��
		for(int j = s; j < e; j++){
			float k = 1.0f - aOld;
			A = samplingTable[j];
			aNew = 1.0f - k*(1.0f - A);
			cAcc.x += k*colorTable[j].x*A;
			cAcc.y += k*colorTable[j].y*A;
			cAcc.z += k*colorTable[j].z*A;
			if(aNew > 0.99f)
				break;
			aOld = aNew;
		}
		int index = s*256 + e;
		pit[index].x = cAcc.x;
		pit[index].y = cAcc.y;
		pit[index].z = cAcc.z;
		pit[index].w = aNew;

		//���簢���� �밢�� /�� �������� �������ϴ�
		aOld = 0, aNew = 0;
		cAcc.x = cAcc.y = cAcc.z = 0;
		for(int j = e; j > s; j--){
			float k = 1.0f - aOld;
			A = samplingTable[j];
			aNew = 1.0f - k*(1.0f - A);
			cAcc.x += k*colorTable[j].x*A;
			cAcc.y += k*colorTable[j].y*A;
			cAcc.z += k*colorTable[j].z*A;
			if(aNew > 0.99f)
				break;
			aOld = aNew;
		}
		index = e*256 + s;
		pit[index].x = cAcc.x;
		pit[index].y = cAcc.y;
		pit[index].z = cAcc.z;
		pit[index].w = aNew;
	}


}


GPUrender::GPUrender(){
	PerspectiveView = false;
	eye[0] = eye[1] = eye[2] = 0;
	float sqr = 1/sqrtf(3);
	L[0] = L[1] = L[2] = sqr;
	up[0] = up[1] = 0;
	up[2] = -1;
	validDir[0] = validDir[1] = validDir[2] = 0;
	zoom = 1.0f;
	resolution = 256;
	pbo = 0;
}

void GPUrender::InitColor(){
	float* gAlphaTable;
	int* gTransparentTable;
	int* gSAT;
	float3* gColorTable;
	bool* gEmptyBlock;
	float time;
	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);	


	hipMalloc((void**)&gSAT, sizeof(int)*257);
	hipMalloc((void**)&gAlphaTable, sizeof(float)*256);
	hipMalloc((void**)&gTransparentTable, sizeof(int)*256);
	hipMalloc((void**)&gColorTable, sizeof(float3)*256);


	hipMemcpy(gAlphaTable, alphaTable, sizeof(float)*256, hipMemcpyHostToDevice);
	hipMemcpy(gColorTable, colorTable, sizeof(float3)*256, hipMemcpyHostToDevice);

	hipEventRecord(start, 0);
	ChangeAlpha<<<1, 256>>>(gAlphaTable, gTransparentTable, gSAT);


	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&time, start, end);
	printf("InitAlpha time = %fms\n", time);

	hipFree(gTransparentTable);

	hipMalloc((void**)&gEmptyBlock, sizeof(bool)*32*32*29);

	hipMemset(gEmptyBlock, 0, 32*32*29*sizeof(bool));
	hipEventRecord(start, 0);
	InitEmptyBlock<<<58, 512>>>(gEmptyBlock, gEmptyBlockMax, gEmptyBlockMin, gSAT);
	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&time, start, end);
	printf("InitEmptyBlock time = %fms\n", time);
	//����޸𸮷� ������������ ������.
	hipMemset(emptyBlock, 0, 32*32*29*sizeof(bool));
	hipMemcpyToSymbol(HIP_SYMBOL(emptyBlock), gEmptyBlock, sizeof(bool)*32*32*29, 0, hipMemcpyDeviceToDevice);

	hipFree(gEmptyBlock);
	hipFree(gSAT);

	hipMalloc((void**)&gPIT, sizeof(float4)*256*256);

	

	hipEventRecord(start, 0);
	InitPreIntegration<<<1, 256>>>(gPIT, gAlphaTable, gColorTable);
	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&time, start, end);
	printf("InitPreintegration time = %fms\n", time);

	hipEventDestroy(start);
	hipEventDestroy(end);
	hipFree(gAlphaTable);
	hipFree(gColorTable);
}
void GPUrender::InitGpuConst(){
	vec_sub(at, eye, dir); //dir ���� ����
	s_product(dir, 1.0f/vec_lenth(dir), dir); //dir�� ����ȭ

	cross_product(up, dir, cross);//cross���� ���� 
	s_product(cross, (256.0f/resolution)*zoom/vec_lenth(cross), cross);//cross���� ����ȭ <- ������ x��ǥ
	//s_product(cross, zoom, cross);//����

	cross_product(dir, cross, u);//u���� ���� 
	s_product(u, (256.0f/resolution)*zoom/vec_lenth(u), u);//u���� ����ȭ <- ������ y��ǥ
	//s_product(u, zoom, u);//����

	if(dir[0] != 0)//x������ 0�� �ƴϸ�
		validDir[0] = 1;
	if(dir[1] != 0)//y������ 0�� �ƴϸ�
		validDir[1] = 1;
	if(dir[2] != 0)//z������ 0�� �ƴϸ�
		validDir[2] = 1;

	//gpu�� ����޸𸮷� ����
	int const_size = sizeof(float)*3;//����޸𸮿� ����� ũ��
	hipMemcpyToSymbol(HIP_SYMBOL(gEye), eye, const_size);
	hipMemcpyToSymbol(HIP_SYMBOL(gDir), dir, const_size);
	hipMemcpyToSymbol(HIP_SYMBOL(gCross), cross, const_size);	
	hipMemcpyToSymbol(HIP_SYMBOL(gU), u, const_size);
	hipMemcpyToSymbol(HIP_SYMBOL(gL), L, const_size);
	hipMemcpyToSymbol(HIP_SYMBOL(gValidDir), validDir, sizeof(int)*3);
	hipMemcpyToSymbol(HIP_SYMBOL(gResolution), &resolution, sizeof(int)*1);
	float backDir[3];
	s_product(dir, 1.3, backDir);
	hipMemcpyToSymbol(HIP_SYMBOL(gBackDir), backDir, const_size);
}
void GPUrender::InitPixelBuffer(){
	glGenBuffers(1, &pbo);//���� ��ü�� �����Ѵ�.
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
	//���ۿ� Ÿ���� ������ (GLenum target,GLuint buffer)
	//Ÿ�ٿ� ���� API�� https://www.EyeBody.org/sdk/docs/man/html/glBindBuffer.xhtml
	//GL_PIXEL_UNPACK_BUFFER�� Texture data source������
	glBufferData(GL_PIXEL_UNPACK_BUFFER, 
					3*resolution*resolution*sizeof(GLubyte), 
					0, 
					GL_STREAM_DRAW);
	//���ε�� ���ۿ� �����ͻ���(�޸� ������)

	//glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);

	hipGraphicsGLRegisterBuffer(&cuda_pbo_resource, 
								pbo, 
								cudaGraphicsMapFlagsNone);
}

void GPUrender::Rendering() {
	unsigned char* gTex;
	//hipError_t result;//�����˻�
	hipEvent_t start, end;

	float time;

	//EyeBody�� �ؽ��ĸ� �����ϴ� �ڵ�
	hipGraphicsMapResources(1, &cuda_pbo_resource, 0);//1�� ����
	hipGraphicsResourceGetMappedPointer((void **)&gTex, NULL, cuda_pbo_resource);//�޸� �����͸� ���´�.(�����Ѵ�)

	//x���� = 8*32 = 256, y���� = 16*16 = 256 => Textureũ��
	//dim3 Dg(8, 16, 1);
	//dim3 Db(32, 16, 1);//32*16 = 512�ִ뾲���� ���
	int block = resolution*resolution/512;

	//�ð����� �ڵ�
	hipEventCreate(&start);
	hipEventCreate(&end);	

	hipEventRecord(start, 0);
	//Ŀ���Լ� ȣ��
	if(PerspectiveView)
		G_Perspective<<<block, 512>>>(gTex, gPIT);
	else
		G_Parallel<<<block, 512>>>(gTex, gPIT);
	
	hipEventRecord(end, 0);

	hipEventSynchronize(end);
	hipEventElapsedTime(&time, start, end);
	hipEventDestroy(start);
	hipEventDestroy(end);
	printf("Renter time = %fms\n", time);
	//�ð����� �ڵ� ��

	//���������ϸ� EyeBody���� �ؽ������
	hipGraphicsUnmapResources(1, &cuda_pbo_resource, 0);

}
void GPUrender::DrawTexture(){
	glClear(GL_COLOR_BUFFER_BIT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexImage2D(GL_TEXTURE_2D, 0, 3, resolution, resolution, 0, GL_RGB, GL_UNSIGNED_BYTE, NULL);
	glEnable(GL_TEXTURE_2D);
	glTexEnvf(GL_TEXTURE_ENV, GL_TEXTURE_ENV_MODE, GL_DECAL);
	glBegin(GL_QUADS);
		glTexCoord2f(0.0, 0.0); glVertex2f(-1.0, -1.0);
		glTexCoord2f(0.0, 1.0); glVertex2f(-1.0, 1.0);
		glTexCoord2f(1.0, 1.0); glVertex2f(1.0, 1.0);
		glTexCoord2f(1.0, 0.0); glVertex2f(1.0, -1.0);
	glEnd();
}

void GPUrender::MouseRotateEye(int x, int y){
	/*	1. eye�� at�� �Ÿ�(A)�� ����Ѵ�.
		2. eye�� ���� cross�� up���� x,y��ŭ �����δ�
		3. ����� eye�� at�� �Ÿ��� A�� �ǵ��� �����.
	*/
	//1��
	float A = sqrtf((eye[0]-at[0])*(eye[0]-at[0]) + (eye[1]-at[1])*(eye[1]-at[1]) + (eye[2]-at[2])*(eye[2]-at[2]));
	//2��
	eye[0] += -x*cross[0] + y*u[0];
	eye[1] += -x*cross[1] + y*u[1];
	eye[2] += -x*cross[2] + y*u[2];

	vec_sub(at, eye, dir); //dir ���� ����
	s_product(dir, 1.0f/vec_lenth(dir), dir); //dir�� ����ȭ
	//3��
	s_product(dir, A, dir);
	vec_sub(at, dir, eye);

	InitGpuConst();
}
void GPUrender::ForwardEye(bool forward){
	if(forward){
		if(PerspectiveView){
			float buf[3];
			s_product(dir, 8, buf);
			vec_add(eye, buf, eye);
		}
		else
			zoom /= 1.1f;
	}
	else{
		if(PerspectiveView){
			float buf[3];
			s_product(dir, 8, buf);
			vec_sub(eye, buf, eye);
		}			
		else
			zoom *= 1.1f;
	}

	printf("eye (%.3f, %.3f, %.3f)\n", eye[0], eye[1], eye[2]);
	printf("dir (%.3f, %.3f, %.3f)\n", dir[0], dir[1], dir[2]);
	InitGpuConst();
}
void GPUrender::ChangeResolution(int n){

	resolution = n;
	InitPixelBuffer();
	InitGpuConst();
}
void GPUrender::ChangeView(bool perspective){
	PerspectiveView = perspective;
}
void GPUrender::InitVolume(unsigned char* Volume, int size[3]){
	volume = Volume;
	volumeSize[0] = size[0];
	volumeSize[1] = size[1];
	volumeSize[2] = size[2];
	at[0] = size[0]/2;
	at[1] = size[1]/2;
	at[2] = size[2]/2;

	glewInit();

	hipMemcpyToSymbol(HIP_SYMBOL(gVolumeSize), volumeSize, sizeof(int)*3);

	int iBlockSize[3] = {volumeSize[0]/8, volumeSize[1]/8, volumeSize[2]/8};
	if(volumeSize[0]%8)
		iBlockSize[0]+=1;
	if(volumeSize[1]%8)
		iBlockSize[1]+=1;
	if(volumeSize[2]%8)
		iBlockSize[2]+=1;

	int fBlockSize[3] = {iBlockSize[0], iBlockSize[1], iBlockSize[2]};
	hipMemcpyToSymbol(HIP_SYMBOL(gBlockSize), fBlockSize, sizeof(int)*3);

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned char>();	
	hipExtent eVolumeSize = make_hipExtent(volumeSize[0], volumeSize[1], volumeSize[2]);

	hipMalloc3DArray(&hipArray, &channelDesc, eVolumeSize, 0);
	hipMemcpy3DParms params = {0};
	params.extent = eVolumeSize;
	params.dstArray = hipArray;
	params.kind = hipMemcpyHostToDevice;
	params.srcPtr = make_hipPitchedPtr((void*)volume, sizeof(unsigned char)*volumeSize[0], volumeSize[0], volumeSize[1]);

	hipMemcpy3D(&params);

	texPtr.filterMode=hipFilterModeLinear;//linear�� texture�� float���� �����ϴ�
	texPtr.addressMode[0]=hipAddressModeWrap;
	texPtr.addressMode[1]=hipAddressModeWrap;
	texPtr.addressMode[2]=hipAddressModeWrap;

	hipBindTextureToArray(texPtr, hipArray, channelDesc); 
	

	
	float time;
	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	
	hipMalloc((void**)&gEmptyBlockMax, sizeof(unsigned char)*iBlockSize[0]*iBlockSize[1]*iBlockSize[2]);
	hipMalloc((void**)&gEmptyBlockMin, sizeof(unsigned char)*iBlockSize[0]*iBlockSize[1]*iBlockSize[2]);
	int block = iBlockSize[0]*iBlockSize[1]*iBlockSize[2]/512;
	if((iBlockSize[0]*iBlockSize[1]*iBlockSize[2])%512)
		block++;
	hipEventRecord(start, 0);
	InitMinMaxEmptyBlock<<<block, 512>>>(gEmptyBlockMax, gEmptyBlockMin);
	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&time, start, end);
	printf("InitMinMaxEmptyBlock time = %fms\n", time);
}
void GPUrender::InitColorTable(float3* ColorTable){
	colorTable = ColorTable;
}
void GPUrender::InitAlphaTable(float* AlphaTable){
	alphaTable = AlphaTable;
}
void GPUrender::EyeBodyCancel(){
	if (pbo) {
		hipGraphicsUnregisterResource(cuda_pbo_resource);//������ ������´�
		glDeleteBuffers(1, &pbo);
	}
	hipUnbindTexture(texPtr); 
	hipFreeArray(hipArray); 
}